
#include <hip/hip_runtime.h>
extern "C" __global__ void blur(const unsigned char *input, unsigned char *output, const unsigned int width, const unsigned int height, const float *kernel, const unsigned int kernelWidth) {

    //const unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    //const unsigned int row = threadIdx.y + blockIdx.y * blockDim.y;
    const unsigned int row = blockIdx.x;
    const unsigned int col = threadIdx.x;

    //printf("col: %d\n", col);
    //printf("row: %d\n", row);

    // blur once with the higher sigma kernel
    if (row < height && col < width) {
        const int half = kernelWidth / 2;
        float blur = 0.0;
        for(int i = -half; i <= half; i++) {
            for(int j = -half; j <= half; j++) {

                const unsigned int y = max(0, min(height - 1, row + i));
                const unsigned int x = max(0, min(width - 1, col + j));

                const float w = kernel[(j + half) + (i + half) * kernelWidth];
                blur += w * input[x + y * width];
            }
        }
        output[col + row * width] = static_cast<unsigned char>(blur);
    }
}
