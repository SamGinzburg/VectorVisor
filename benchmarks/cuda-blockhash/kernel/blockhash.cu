
#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C" __global__ void blockhash(unsigned char *input_img, unsigned int *blocks, const unsigned int block_width, const unsigned int block_height, const unsigned int bits) {

    // x, y pixel values
    const unsigned int x = blockIdx.x;
    const unsigned int y = threadIdx.y;
    // bits == the number of blocks to divide by horizontally/vertically
    // If img dims are 256x256 with a block size of 8, then bits==32

    // fast blockhash alg, assume width % 4 == 0 and height % 4 == 0

    // Blocks by default are 8 pixels x 8 pixels in size for our benchmark

    // For each pixel
    // 1) Compute the block value
    // 1.1) Check alpha channel for pixel, if alpha==0 then value = 765
    // 1.2) Else, value = sum(px) (RGB channels) for each value in a block
    // 1.3) Assign block value

    int64_t value = 0;
    for (uint64_t blocky = 0; blocky < block_height; blocky++) {
        for (uint64_t blockx = 0; blockx < block_width; blockx++) {
            uint64_t block_idx = ((y * block_height + blocky) * block_width + (x * block_width + blockx)) * 4;
            uint32_t alpha = input_img[block_idx+3];
            if (alpha == 0) {
                value += 765;
            } else {
                value += input_img[block_idx] + input_img[block_idx+1] + input_img[block_idx+2];
            }
        }
    }

    blocks[y * bits + x] = value;
}
